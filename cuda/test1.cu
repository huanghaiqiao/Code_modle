#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  printf("%d \n",N);
  float *x, *y, *z;
//   cudaProfilerStart();

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMalloc(&z, N*sizeof(float)); // only for GPU
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
//   cudaDeviceReset();
//   cudaProfilerStop();

  return 0;
}

